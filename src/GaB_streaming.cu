#include "hip/hip_runtime.h"
/* ###########################################################################################################################
## Organization         : The University of Arizona
##                      :
## File name            : GaB.cu
## Language             : CUDA C (ANSI)
## Short description    : Parallel Based Gallager-B Hard decision Bit-Flipping algorithm
##                      :
##                      :
##                      :
## History              : Modified 19/01/2016, Created by Burak UNAL
##                      : Modified Spring 2022, Parallellized by Christopher Brown and Jared Causey
##                      :
## COPYRIGHT            : burak@email.arizona.edu
## ######################################################################################################################## */
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stdio.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include "kernels.cu"

#define arrondi(x) ((ceil(x)-x)<(x-floor(x))?(int)ceil(x):(int)floor(x))
#define min(x,y) ((x)<(y)?(x):(y))
#define signf(x) ((x)>=0?0:1)
#define	max(x,y) ((x)<(y)?(y):(x))
#define SQR(A) ((A)*(A))
#define BPSK(x) (1-2*(x))
#define PI 3.1415926536


//#####################################################################################################
void DataPassGB(int *VtoC,int *CtoV,int *Receivedword,int *Interleaver,int *ColumnDegree,int N,int* numBcol)
{
	int t,numB,n,buf;
	int Global;
	numB=0;
	for (n=0;n<N;n++)
	{
		//Global=(Amplitude)*(1-2*ReceivedSymbol[n]);
		Global=(1-2*Receivedword[n]); 
		//Global=(1-2*(Decide[n] + Receivedword[n])); //Decide[n]^Receivedword[n];
		for (t=0;t<ColumnDegree[n];t++) Global+=(-2)*CtoV[Interleaver[numB+t]]+1;

		for (t=0;t<ColumnDegree[n];t++)
		{
		  buf=Global-((-2)*CtoV[Interleaver[numB+t]]+1);
		  if (buf<0)  VtoC[Interleaver[numB+t]]= 1; //else VtoC[Interleaver[numB+t]]= 1;
		  else if (buf>0) VtoC[Interleaver[numB+t]]= 0; //else VtoC[Interleaver[numB+t]]= 1;
		  else  VtoC[Interleaver[numB+t]]=Receivedword[n];
		}
		numB=numB+ColumnDegree[n];
	}
}
//#####################################################################################################
void DataPassGBIter0(int *VtoC,int *CtoV,int *Receivedword,int *Interleaver,int *ColumnDegree,int N,int* numBcol)
{
	int t,numB,n;
	numB=0;
	for (n=0;n<N;n++)
	{
		for (t=0;t<ColumnDegree[n];t++)     VtoC[Interleaver[numB+t]]=Receivedword[n];
		numB=numB+ColumnDegree[n];
	}
}
//##################################################################################################
void CheckPassGB(int *CtoV,int *VtoC,int M,int* numBrow,int *RowDegree)
{
   int t,numB=0,m,signe;
   for (m=0;m<M;m++)
   {
		signe=0;for (t=0;t<RowDegree[m];t++) signe^=VtoC[numB+t];
	    for (t=0;t<RowDegree[m];t++) 	CtoV[numB+t]=signe^VtoC[numB+t];
		numB=numB+RowDegree[m];
   }
}
//#####################################################################################################
void APP_GB(int *Decide,int *CtoV,int *Receivedword,int *Interleaver,int *ColumnDegree,int N,int M,int* numBcol)
{
   	int t,numB,n;
	int Global;
	numB=0;
	for (n=0;n<N;n++)
	{
		Global=(1-2*Receivedword[n]);
		for (t=0;t<ColumnDegree[n];t++) Global+=(-2)*CtoV[Interleaver[numB+t]]+1;
        if(Global>0) Decide[n]= 0;
        else if (Global<0) Decide[n]= 1;
        else  Decide[n]=Receivedword[n];
		numB=numB+ColumnDegree[n];
	}
}
//#####################################################################################################
int ComputeSyndrome(int *Decide,int **Mat,int *RowDegree,int M)
{
	int Synd,k,l;

	for (k=0;k<M;k++)
	{
		Synd=0;
		for (l=0;l<RowDegree[k];l++) Synd=Synd^Decide[Mat[k][l]];
		if (Synd==1) break;
	}
	return(1-Synd);
}
//#####################################################################################################
int GaussianElimination_MRB(int *Perm,int **MatOut,int **Mat,int M,int N)
{
	int k,n,m,m1,buf,ind,indColumn,nb,*Index,dep,Rank;

	Index=(int *)calloc(N,sizeof(int));

	// Triangularization
	indColumn=0;nb=0;dep=0;
	for (m=0;m<M;m++)
	{
		if (indColumn==N) { dep=M-m; break; }

		for (ind=m;ind<M;ind++) { if (Mat[ind][indColumn]!=0) break; }
		// If a "1" is found on the column, permutation of rows
		if (ind<M)
		{
			for (n=indColumn;n<N;n++) { buf=Mat[m][n]; Mat[m][n]=Mat[ind][n]; Mat[ind][n]=buf; }
		// bottom of the column ==> 0
			for (m1=m+1;m1<M;m1++)
			{
				if (Mat[m1][indColumn]==1) { for (n=indColumn;n<N;n++) Mat[m1][n]=Mat[m1][n]^Mat[m][n]; }
			}
			Perm[m]=indColumn;
		}
		// else we "mark" the column.
		else { Index[nb++]=indColumn; m--; }

		indColumn++;
	}

	Rank=M-dep;

	for (n=0;n<nb;n++) Perm[Rank+n]=Index[n];

	// Permutation of the matrix
	for (m=0;m<M;m++) { for (n=0;n<N;n++) MatOut[m][n]=Mat[m][Perm[n]]; }

	// Diagonalization
	for (m=0;m<(Rank-1);m++)
	{
		for (n=m+1;n<Rank;n++)
		{
			if (MatOut[m][n]==1) { for (k=n;k<N;k++) MatOut[m][k]=MatOut[n][k]^MatOut[m][k]; }
		}
	}
	free(Index);
	return(Rank);
}

//#####################################################################################################
int main(int argc, char * argv[])
{
  // Variables Declaration
  FILE *f;
  int Graine,NbIter,nbtestedframes,NBframes;
  float alpha_max, alpha_min,alpha_step,alpha,NbMonteCarlo;
  // ----------------------------------------------------
  // lecture des param de la ligne de commande
  // ----------------------------------------------------
  char *FileName,*FileMatrix,*FileResult;
  FileName=(char *)malloc(200);
  FileMatrix=(char *)malloc(200);
  FileResult=(char *)malloc(200);

  strcpy(FileMatrix,argv[1]); 	// Matrix file
  strcpy(FileResult,argv[2]); 	// Results file

  //--------------Simulation input for GaB BF-------------------------
  NbMonteCarlo=100000000000;	    // Maximum nb of codewords sent
  NbIter=100; 	            // Maximum nb of iterations
  alpha= 0.01;              // Channel probability of error
  NBframes=100;	            // Simulation stops when NBframes in error
  Graine=1;		            // Seed Initialization for Multiple Simulations

    // brkunl
  alpha_max= 0.0600;		    //Channel Crossover Probability Max and Min
  alpha_min= 0.0200;
  alpha_step=0.0100;


  // ----------------------------------------------------
  // Load Matrix
  // ----------------------------------------------------
  int *ColumnDegree,*RowDegree,**Mat;
  int M,N,m,n,k;
  strcpy(FileName,FileMatrix);strcat(FileName,"_size");
  f=fopen(FileName,"r");fscanf(f,"%d",&M);fscanf(f,"%d",&N);
  ColumnDegree=(int *)calloc(N,sizeof(int));
  RowDegree=(int *)calloc(M,sizeof(int));fclose(f);
  strcpy(FileName,FileMatrix);strcat(FileName,"_RowDegree");
  f=fopen(FileName,"r");for (m=0;m<M;m++) fscanf(f,"%d",&RowDegree[m]);fclose(f);
  Mat=(int **)calloc(M,sizeof(int *));for (m=0;m<M;m++) Mat[m]=(int *)calloc(RowDegree[m],sizeof(int));
  strcpy(FileName,FileMatrix);
  f=fopen(FileName,"r");for (m=0;m<M;m++) { for (k=0;k<RowDegree[m];k++) fscanf(f,"%d",&Mat[m][k]); }fclose(f);
  for (m=0;m<M;m++) { for (k=0;k<RowDegree[m];k++) ColumnDegree[Mat[m][k]]++; }

  printf("Matrix Loaded \n");

  // ----------------------------------------------------
  // Build Graph
  // ----------------------------------------------------
  int NbBranch,**NtoB,*Interleaver,*ind,numColumn,numBranch;
  NbBranch=0; for (m=0;m<M;m++) NbBranch=NbBranch+RowDegree[m];
  NtoB=(int **)calloc(N,sizeof(int *)); for (n=0;n<N;n++) NtoB[n]=(int *)calloc(ColumnDegree[n],sizeof(int));
  Interleaver=(int *)calloc(NbBranch,sizeof(int));
  ind=(int *)calloc(N,sizeof(int));
  numBranch=0;for (m=0;m<M;m++) { for (k=0;k<RowDegree[m];k++) { numColumn=Mat[m][k]; NtoB[numColumn][ind[numColumn]++]=numBranch++; } }
  free(ind);
  numBranch=0;for (n=0;n<N;n++) { for (k=0;k<ColumnDegree[n];k++) Interleaver[numBranch++]=NtoB[n][k]; }

  printf("Graph Build \n");

  // ----------------------------------------------------
  // Decoder
  // ----------------------------------------------------
  int *CtoV,*VtoC,*Codeword,*Receivedword,*Decide,*U,l,*numBrow,*numBcol;
  int iter;
  CtoV=(int *)calloc(NbBranch,sizeof(int));
  VtoC=(int *)calloc(NbBranch,sizeof(int));
  Codeword=(int *)calloc(N,sizeof(int));
  Receivedword=(int *)calloc(N,sizeof(int));
  Decide=(int *)calloc(N,sizeof(int));
  U=(int *)calloc(N,sizeof(int));
  srand48(time(0)+Graine*31+113);

  //precompute numB values
  numBrow=(int *)calloc(M,sizeof(int));
 	int numB=0;
  for (m=0;m<M;m++)
  {
    // if(m == M-1){
    //   printf("numBrow[%d]= %d\n",m,numB);
    // }
		numBrow[m] = numB;
    // printf("%d\n", numB);
    numB=numB+RowDegree[m];
  }
    // exit(0);


  numBcol=(int *)calloc(N,sizeof(int));
	numB=0;
	for (n=0;n<N;n++)
	{
    // if(n == N-1){
    //   printf("numBcol[%d]= %d\n",n,numB);
    // }
		numBcol[n] = numB;
    // printf("%d\n", numB);
		numB=numB+ColumnDegree[n];
	}

  // ----------------------------------------------------
  // Allocate and fill GPU Data for Matrix and Decoder
  // ----------------------------------------------------
  int *device_ColumnDegree,*device_RowDegree,**device_Mat,*device_Interleaver,*device_numBrow,*device_numBcol;
  
  // Initialize and Fill Matrix and Degree Arrays on Device (Should never be modified)
  hipMalloc((void **)&device_Mat, M * sizeof(int*));
  int** temp_i_ptrs = (int**) malloc(M * sizeof(int*));
  for (m=0;m<M;m++){
    hipMalloc((void**)&temp_i_ptrs[m], RowDegree[m] * sizeof(int));
    hipMemcpy(temp_i_ptrs[m], Mat[m], RowDegree[m] * sizeof(int), hipMemcpyHostToDevice);
  }
  hipMemcpy(device_Mat, temp_i_ptrs, sizeof(int*) * M, hipMemcpyHostToDevice);

  // for(int a=0;a<M;a++){for(int b=0;b<RowDegree[a];b++){printf("%d\n",Mat[a][b]);}}

  hipMalloc((void **)&device_RowDegree, M * sizeof(int));
  hipMemcpy(device_RowDegree, RowDegree, M * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void **)&device_ColumnDegree, N * sizeof(int));
  hipMemcpy(device_ColumnDegree, ColumnDegree, N * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void **)&device_Interleaver, NbBranch * sizeof(int));
  hipMemcpy(device_Interleaver, Interleaver, NbBranch * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void **)&device_numBrow, M * sizeof(int));
  hipMemcpy(device_numBrow, numBrow, M * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void **)&device_numBcol, N * sizeof(int));
  hipMemcpy(device_numBcol, numBcol, N * sizeof(int), hipMemcpyHostToDevice);

  int *device_CtoV,*device_VtoC,*device_Codeword,*device_Receivedword,*device_Decide,*device_IsCodeword;

  // Initialize GaB node connections and Codeword Arrays on Device
  hipMalloc((void **)&device_CtoV, NbBranch * sizeof(int));
  hipMemset((void **)&device_CtoV, 0, NbBranch * sizeof(int));
  
  hipMalloc((void **)&device_VtoC, NbBranch * sizeof(int));
  hipMemset((void **)&device_VtoC, 0, NbBranch * sizeof(int));
  
  const int code_word_size = N;//8 * sizeof(int);
  hipMalloc((void **)&device_Codeword, N * sizeof(int));
  hipMemset((void **)&device_Codeword, 0, N * sizeof(int));  
  
  hipMalloc((void **)&device_Receivedword, code_word_size * sizeof(int));
  hipMemset((void **)&device_Receivedword, 0, code_word_size * sizeof(int));   
  
  hipMalloc((void **)&device_Decide, code_word_size * sizeof(int));
  hipMemset((void **)&device_Decide, 0, code_word_size * sizeof(int));   

  hipMalloc((void **)&device_IsCodeword, sizeof(int));
  
  // Set Up GPU Kernel Dimensions
  dim3 blockDim(M),gridDim(32);

  // dim3 blockDim,gridDim(32);
  // if(M > N){
  //   blockDim = (M/2);
  // }
  // else{
  //   blockDim = (N/2);
  // }

  // ----------------------------------------------------
  // Initialize Timing Structures
  // ----------------------------------------------------
  hipEvent_t astartEvent, astopEvent;
  float aelapsedTime;
  hipEventCreate(&astartEvent);
  hipEventCreate(&astopEvent);

  // ----------------------------------------------------
  // Gaussian Elimination for the Encoding Matrix (Full Representation)
  // ----------------------------------------------------
  int **MatFull,**MatG,*PermG;
  int rank;
  MatG=(int **)calloc(M,sizeof(int *));for (m=0;m<M;m++) MatG[m]=(int *)calloc(N,sizeof(int));
  MatFull=(int **)calloc(M,sizeof(int *));for (m=0;m<M;m++) MatFull[m]=(int *)calloc(N,sizeof(int));
  PermG=(int *)calloc(N,sizeof(int)); for (n=0;n<N;n++) PermG[n]=n;
  for (m=0;m<M;m++) { for (k=0;k<RowDegree[m];k++) { MatFull[m][Mat[m][k]]=1; } }
  rank=GaussianElimination_MRB(PermG,MatG,MatFull,M,N);
  //for (m=0;m<N;m++) printf("%d\t",PermG[m]);printf("\n");

  // Variables for Statistics
  int IsCodeword,nb;
  int NiterMoy,NiterMax;
  int Dmin;
  int NbTotalErrors,NbBitError;
  int NbUnDetectedErrors,NbError;
  float timeAverage;


  strcpy(FileName,FileResult);
  f=fopen(FileName,"w");
  fprintf(f,"-------------------------Gallager B--------------------------------------------------\n");
  fprintf(f,"alpha\t\tNbEr(BER)\t\tNbFer(FER)\t\tNbtested\t\tIterAver(Itermax)\t\tNbUndec(Dmin)\t\tTimePerFrame\n");

  printf("-------------------------Gallager B--------------------------------------------------\n");
  printf("alpha\t\tNbEr(BER)\t\tNbFer(FER)\t\tNbtested\t\tIterAver(Itermax)\t\tNbUndec(Dmin)\t\tTimePerFrame\n");

  // Set up CUDA stream objects
  char* temp;
  const int num_streams = strtol(argv[3],&temp,10);
  printf("Creating %d streams\n",num_streams);
  hipStream_t* pStreams = (hipStream_t*)malloc(num_streams * sizeof(hipStream_t));
  for (int i = 0; i < num_streams; i++)
    hipStreamCreate(&(pStreams[i]));

  for(alpha=alpha_max;alpha>=alpha_min;alpha-=alpha_step) {

  NiterMoy=0;NiterMax=0;
  Dmin=1e5;
  NbTotalErrors=0;NbBitError=0;
  NbUnDetectedErrors=0;NbError=0;
  timeAverage=0.0;
  //--------------------------------------------------------------
  for (nb=0,nbtestedframes=0;nb<NbMonteCarlo;nb++)
  {
  //encoding
  for (k=0;k<rank;k++) U[k]=0;
	for (k=rank;k<N;k++) U[k]=floor(drand48()*2);
	for (k=rank-1;k>=0;k--) { for (l=k+1;l<N;l++) U[k]=U[k]^(MatG[k][l]*U[l]); }
	for (k=0;k<N;k++) Codeword[PermG[k]]=U[k];

	// All zero codeword
	//for (n=0;n<N;n++) { Codeword[n]=0; }

  // Add Noise
  for (n=0;n<N;n++)  if (drand48()<alpha) Receivedword[n]=1-Codeword[n]; else Receivedword[n]=Codeword[n];
  //============================================================================
 	// Decoder
	//============================================================================
  hipEventRecord(astartEvent, 0);
  if(argc == 4){ //parallel
    //printf("decoding\n");
    for (int stream_cnt = 0; stream_cnt < num_streams; stream_cnt++)
    {
      // Copy Received Word to the GPU
      hipMemcpyAsync(device_Decide, Receivedword/*+(code_word_size*stream_cnt)*/, code_word_size * sizeof(int), hipMemcpyHostToDevice, pStreams[stream_cnt]);
      hipMemcpyAsync(device_Receivedword, Receivedword/*+(code_word_size*stream_cnt)*/, code_word_size * sizeof(int), hipMemcpyHostToDevice, pStreams[stream_cnt]);
    }
    for (int stream_cnt = 0; stream_cnt < num_streams; stream_cnt++)
    {
      for (iter=0;iter<NbIter;iter++)
      {
        // Reset IsCodeword
        hipMemsetAsync(device_IsCodeword, 1, sizeof(int), pStreams[stream_cnt]);
        // Call Decode
        global_decode<<<gridDim,blockDim,0,pStreams[stream_cnt]>>>(device_VtoC,device_CtoV,device_Mat,device_RowDegree,device_ColumnDegree,
                                            device_Decide,device_Receivedword,device_Interleaver,M,N,
                                            device_numBrow,device_numBcol,iter,device_IsCodeword);
        //Retreive IsCodeWord
        hipMemcpyAsync(&IsCodeword,device_IsCodeword, sizeof(int), hipMemcpyDeviceToHost,pStreams[stream_cnt]);
        if (IsCodeword)
          break;
      }
      // Stream 1
      hipMemcpyAsync(Decide, device_Decide/*+(code_word_size*stream_cnt)*/, code_word_size * sizeof(int), hipMemcpyDeviceToHost,pStreams[stream_cnt]);
    }
  }
  else{ //serial
    // REPLACE THE CODE BELOW WITH CUDA KERNEL CALLS -------------------------------------------------
    for (k=0;k<NbBranch;k++) {CtoV[k]=0;}

    for (k=0;k<N;k++) Decide[k]=Receivedword[k];

    for (iter=0;iter<NbIter;iter++){
        if(iter==0){
          DataPassGBIter0(VtoC,CtoV,Receivedword,Interleaver,ColumnDegree,N,numBcol);
        }
        else{
          DataPassGB(VtoC,CtoV,Receivedword,Interleaver,ColumnDegree,N,numBcol);
        }
        
        CheckPassGB(CtoV,VtoC,M,numBrow,RowDegree);
        APP_GB(Decide,CtoV,Receivedword,Interleaver,ColumnDegree,N,M,numBcol);
        
        IsCodeword=ComputeSyndrome(Decide,Mat,RowDegree,M);
        if (IsCodeword){
          break;
        } 
    }
    // -----------------------------------------------------------------------------------------------
  }
  hipEventRecord(astopEvent, 0);
  hipEventSynchronize(astopEvent);
  hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
  timeAverage += aelapsedTime;
	//============================================================================
  	// Compute Statistics
	//============================================================================
  nbtestedframes++;
	NbError=0;for (k=0;k<N;k++)  if (Decide[k]!=Codeword[k]) NbError++;
	NbBitError=NbBitError+NbError;
	
  // Case Divergence
	if (!IsCodeword)
	{
	  NiterMoy=NiterMoy+NbIter;
		NbTotalErrors++;
	}
	
  // Case Convergence to Right Codeword
	if ((IsCodeword)&&(NbError==0)) { NiterMax=max(NiterMax,iter+1); NiterMoy=NiterMoy+(iter+1); }
	
  // Case Convergence to Wrong Codeword
	if ((IsCodeword)&&(NbError!=0))
	{
	  NiterMax=max(NiterMax,iter+1); NiterMoy=NiterMoy+(iter+1);
	  NbTotalErrors++; NbUnDetectedErrors++;
	  Dmin=min(Dmin,NbError);
	}

	// Stopping Criterion
	if (NbTotalErrors==NBframes) break;
  }

  float timeAveragePerNb = timeAverage/nbtestedframes;
  
  printf("%1.5f\t\t",alpha);
  printf("%10d (%1.16f)\t\t",NbBitError,(float)NbBitError/N/nbtestedframes);
  printf("%4d (%1.16f)\t\t",NbTotalErrors,(float)NbTotalErrors/nbtestedframes);
  printf("%10d\t\t",nbtestedframes);
  printf("%1.2f(%d)\t\t",(float)NiterMoy/nbtestedframes,NiterMax);
  printf("%d(%d)\t\t",NbUnDetectedErrors,Dmin);
  printf("%f\n",timeAveragePerNb);


  fprintf(f,"%1.5f\t\t",alpha);
  fprintf(f,"%10d (%1.8f)\t\t",NbBitError,(float)NbBitError/N/nbtestedframes);
  fprintf(f,"%4d (%1.8f)\t\t",NbTotalErrors,(float)NbTotalErrors/nbtestedframes);
  fprintf(f,"%10d\t\t",nbtestedframes);
  fprintf(f,"%1.2f(%d)\t\t",(float)NiterMoy/nbtestedframes,NiterMax);
  fprintf(f,"%d(%d)\t\t",NbUnDetectedErrors,Dmin);
  fprintf(f,"%f\n",timeAveragePerNb);

}

for (int i = 0; i < num_streams; i++)
  hipStreamDestroy(pStreams[i]);
free(pStreams);

// Free up GPU memory
hipFree(device_Mat);
hipFree(device_RowDegree);
hipFree(device_ColumnDegree);
hipFree(device_Interleaver);
hipFree(device_numBrow);
hipFree(device_numBcol);
hipFree(device_CtoV);
hipFree(device_VtoC);
hipFree(device_Codeword);
hipFree(device_Receivedword);
hipFree(device_Decide);
hipFree(device_IsCodeword);

fclose(f);
return(0);
}
